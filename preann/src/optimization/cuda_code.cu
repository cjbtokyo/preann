#include "hip/hip_runtime.h"
#include "cuda_code.h"

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		printf("Cuda error: %s : %s.\n", msg, hipGetErrorString(err));
		exit( EXIT_FAILURE);
	}
}

/// ACTIVATION
__device__
float Func(float number, FunctionType functionType)
{
	switch (functionType) {

	case FT_BINARY_STEP:
		if (number > 0) {
			return 1;
		} else {
			return 0;
		}
	case FT_BIPOLAR_STEP:
		if (number > 0) {
			return 1;
		} else {
			return -1;
		}
	case SIGMOID:
		return 1.0f / (1.0f - exp(-number));
	case FT_BIPOLAR_SIGMOID:
		return -1.0f + (2.0f / (1.0f + exp(-number)));
	case FT_HYPERBOLIC_TANGENT:
		return tanh(number);
	case FT_IDENTITY:
	default:
		return number;
	}
}

__global__
void activation_float_kernel(float* results, float* output, unsigned output_sz,
		FunctionType functionType)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < output_sz)
		output[idx] = Func(results[idx], functionType);
}

__global__
void activation_bit_kernel(float* results, unsigned* output, unsigned output_sz)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned offset = idx * BITS_PER_UNSIGNED;

	if (output_sz > offset) {

		unsigned toRead = min(BITS_PER_UNSIGNED, output_sz - offset);
		unsigned threadOutput = 0;
		unsigned mask = 0x80000000;

		for (unsigned i = 0; i < toRead; i++) {
			if (results[offset + i] > 0) {
				threadOutput |= mask;
			} else {
				threadOutput &= ~mask;
			}
			mask >>= 1;
		}
		output[idx] = threadOutput;
	}
}

extern "C" void cuda_activation(void* data, unsigned size, BufferType bufferType, float* results, FunctionType functionType, unsigned block_size)
{
	unsigned grid_size;

	switch (bufferType){
	case BT_BYTE:
		{
			std::string error = "cuda_activation is not implemented for BufferType BYTE.";
			throw error;
		}
	case BT_FLOAT:
		{
			grid_size = ((size - 1) / block_size) + 1;
			activation_float_kernel<<< grid_size, block_size >>>(results, (float*)data, size, functionType);
		}
		break;
	case BT_BIT:
	case BT_SIGN:
		{
			grid_size = ((size - 1) / (block_size * BITS_PER_UNSIGNED)) + 1;
			activation_bit_kernel<<< grid_size, block_size >>>(results, (unsigned*)data, size);
		}
		break;
	}
	checkCUDAError("activation");
}

// MEMORY MANAGEMENT

extern "C" void* cuda_malloc(unsigned byteSize)
{
	void* ptr;
	hipMalloc((void**) &(ptr), byteSize);

	checkCUDAError("malloc");
	return ptr;
}

extern "C" void cuda_free(void* d_ptr)
{
	hipFree(d_ptr);
	checkCUDAError("free");
}

extern "C" void cuda_copyToDevice(void* d_dest, void* h_src, unsigned count)
{
	hipMemcpy(d_dest, h_src, count, hipMemcpyHostToDevice);
	checkCUDAError("copyToDevice");
}

extern "C" void cuda_copyToHost(void* h_dest, void* d_src, unsigned count)
{
	hipMemcpy(h_dest, d_src, count, hipMemcpyDeviceToHost);
	checkCUDAError("copyToHost");
}

// INITIALIZATION

template <class bufferType>
__global__
void SetValueToAnArrayKernel(bufferType* data, unsigned size, bufferType value)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		data[idx] = value;
}

extern "C" void cuda_setZero(void* data, unsigned byteSize, BufferType bufferType, unsigned block_size)
{
	unsigned grid_size;
	unsigned size;

	switch (bufferType){
	case BT_BYTE:
		size = byteSize / sizeof(unsigned char);
		grid_size = ((size - 1) / block_size) + 1;
		SetValueToAnArrayKernel<unsigned char><<< grid_size, block_size >>>((unsigned char*)data, size, (unsigned char)0);
		break;
	case BT_FLOAT:
		size = byteSize / sizeof(float);
		grid_size = ((size - 1) / block_size) + 1;
		SetValueToAnArrayKernel<float><<< grid_size, block_size >>>((float*)data, size, 0);
		break;
	case BT_BIT:
	case BT_SIGN:
		hipMemset(data, 0, byteSize);
		break;
	}
}

// GENETIC OPERATORS

template <class type>
__global__
void crossoverKernel(type* buffer1, type* buffer2, unsigned* bitBuffer, unsigned size)
{
	unsigned weighPos = (blockIdx.x * blockDim.x * BITS_PER_UNSIGNED) + threadIdx.x;
	unsigned maxPosForThisBlock = min ( (blockIdx.x + 1) * blockDim.x * BITS_PER_UNSIGNED,
										size);
	unsigned bitsForTheThread, mask;
	if (weighPos < maxPosForThisBlock) {
		bitsForTheThread = bitBuffer[(blockIdx.x * blockDim.x) + threadIdx.x];
		mask = 0x80000000;
	}
	__syncthreads();
	while (weighPos < maxPosForThisBlock){
		if (mask & bitsForTheThread){
			type aux = buffer1[weighPos];
			buffer1[weighPos] = buffer2[weighPos];
			buffer2[weighPos] = aux;
		}
		weighPos += blockDim.x;
		mask >>= 1;
	}
}

extern "C"
void cuda_crossover(void* buffer1, void* buffer2, unsigned* bitBuffer, unsigned size, BufferType bufferType,unsigned block_size)
{
	unsigned grid_size = ((size - 1)/(block_size * BITS_PER_UNSIGNED)) + 1;

	switch (bufferType){
        case BT_BYTE:
		crossoverKernel<unsigned char><<< grid_size, block_size >>>
				((unsigned char*)buffer1, (unsigned char*)buffer2, (unsigned*)bitBuffer, size);

        break;
    case BT_FLOAT:
    	crossoverKernel<float><<< grid_size, block_size >>>
				((float*)buffer1, (float*)buffer2, (unsigned*)bitBuffer, size);
		break;
	case BT_BIT:
	case BT_SIGN:
		{
		std::string error = "cuda_crossover is not implemented for BufferType BIT nor SIGN.";
		throw error;
		}
	}
}

//TODO CU es necesario usar un kernel para esto ??
__global__
void resetFloatKernel(float* buffer, unsigned pos)
{
	if (threadIdx.x == 0){
		buffer[pos] = 0;
	}
}

__global__
void resetByteKernel(unsigned char* buffer, unsigned pos)
{
	if (threadIdx.x == 0){
		buffer[pos] = 128;
	}
}

__global__
void mutateFloatKernel(float* buffer, unsigned pos, float mutation)
{
	if (threadIdx.x == 0){
		buffer[pos] += mutation;
	}
}

__global__
void mutateByteKernel(unsigned char* buffer, unsigned pos, int mutation)
{
	if (threadIdx.x == 0){
		int result = mutation + buffer[pos];
		if (result <= 0){
			buffer[pos] = 0;
		}
		else if (result >= 255) {
			buffer[pos] = 255;
		}
		else {
			buffer[pos] = (unsigned char)result;
		}
	}
}

extern "C" void cuda_mutate(void* buffer, unsigned pos, float mutation, BufferType bufferType)
{
	switch (bufferType){
	case BT_BYTE:
		mutateByteKernel<<< 1, 8 >>>((unsigned char*)buffer, pos, (int)mutation);
		break;
	case BT_FLOAT:
		mutateFloatKernel<<< 1, 8 >>>((float*)buffer, pos, mutation);
		break;
	case BT_BIT:
	case BT_SIGN:
		{
		std::string error = "cuda_mutate is not implemented for BufferType BIT nor SIGN.";
		throw error;
		}
	}
}

extern "C" void cuda_reset(void* buffer, unsigned pos, BufferType bufferType)
{
	switch (bufferType){
	case BT_BYTE:
		resetByteKernel<<< 1, 8 >>>((unsigned char*)buffer, pos);
		break;
	case BT_FLOAT:
		resetFloatKernel<<< 1, 8 >>>((float*)buffer, pos);
		break;
	case BT_BIT:
	case BT_SIGN:
		{
		std::string error = "cuda_reset is not implemented for BufferType BIT nor SIGN.";
		throw error;
		}
	}
}

// CL_LAYER CALCULATION

__global__
void SumFloatsConnectionsKernel(float* inputs, unsigned input_size, unsigned output_size, float* weighs, float* results)
{
	extern __shared__ float sdata[];

	unsigned outputNeuron = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned weighsOffset = outputNeuron * input_size;
	float result = 0;

	unsigned pos = threadIdx.x;
	while (pos < input_size) {

		sdata[pos] = inputs[pos];
		pos += blockDim.x;
	}
	__syncthreads();

	if (outputNeuron < output_size) {

		//////////////////////////
		for (unsigned i = 0; i < input_size; i++) {
			result += sdata[i] * weighs[weighsOffset + i];
			//printf(" peso %f ", weighs[weighsOffset + i]);
		}
		/////TODO TR OTRA OPCION
	/*	if (blockDim.x <= input_size){
			unsigned pos = tid;
			while (pos < input_size){
				result += sdata[pos] * weighs[weighsOffset + pos];
				++pos;
			}
			pos = 0;
			while (pos < tid){
				result += sdata[pos] * weighs[weighsOffset + pos];
				++pos;
			}
		} else {
			unsigned pos = tid;
			while (pos < input_size){
				result += sdata[pos] * weighs[weighsOffset + pos];
				++pos;
			}
			unsigned newMax = min(tid, input_size);
			pos = 0;
			while (pos < newMax){
				result += sdata[pos] * weighs[weighsOffset + pos];
				++pos;
			}
		}*/
		/////////////
		results[outputNeuron] += result;
	}
}

template <BufferType inputType>
__global__
void SumBitsConnectionsKernel(unsigned* inputs, unsigned input_size, unsigned output_size, unsigned char* weighs, float* results)
{
	extern __shared__ unsigned shared_inputs[];

	unsigned tid = threadIdx.x;
	unsigned input_blocks_to_read = ((input_size - 1) / BITS_PER_UNSIGNED) + 1;
	unsigned readingLoops = ((input_blocks_to_read - 1) / blockDim.x) + 1;

	unsigned pos = tid;

	for (unsigned i=0; i < readingLoops; i++) {
		if (pos < input_blocks_to_read) {
			shared_inputs[pos] = inputs[pos];
		}
		pos += blockDim.x;
	}
	__syncthreads();

	unsigned outputNeuron = blockIdx.x*blockDim.x + threadIdx.x;
	if (outputNeuron < output_size) {

		float result = 0;
		unsigned weighsOffset = (outputNeuron * input_size);

		for (unsigned i=0; i < input_blocks_to_read; i++) {

			//TODO TCC check performance penalty (this is just for BT_SIGN)
			unsigned maxBits = min(BITS_PER_UNSIGNED, input_size - (i * BITS_PER_UNSIGNED));

			unsigned input_block = shared_inputs[i];
			unsigned mask = 0x80000000;
			for (unsigned j=0; j < maxBits; j++) {

				if (input_block & mask) {
					result += weighs[weighsOffset] - 128;
				} else {
					if (inputType == BT_SIGN) {
						result += 128 - weighs[weighsOffset];
					}
				}
				++weighsOffset;
				mask >>= 1;
			}
		}
		results[outputNeuron] += result;
	}
}

__global__
void SumFloatsInvertedConnectionsKernel(float* inputs, unsigned input_size,
		float* weighs, float* results, unsigned output_size)
{
	extern __shared__ float sdata[];

	unsigned input_pos = threadIdx.x;
	while (input_pos < input_size) {

		sdata[input_pos] = inputs[input_pos];
		input_pos += blockDim.x;
	}
	__syncthreads();

	unsigned output_pos = blockIdx.x * blockDim.x + threadIdx.x;
	float result = 0;

	if (output_pos < output_size) {

		for (unsigned i = 0; i < input_size; i++) {
			result += sdata[i] * weighs[output_pos + (i * output_size)];
		}
		results[output_pos] += result;
	}
}

template <BufferType inputType>
__global__
void SumBitsInvertedConnectionsKernel(unsigned* inputs, unsigned input_size, unsigned output_size, unsigned char* weighs, float* results)
{
	extern __shared__ unsigned shared_inputs[];

	unsigned tid = threadIdx.x;
	unsigned input_blocks_to_read = ((input_size - 1) / BITS_PER_UNSIGNED) + 1;
	unsigned readingLoops = ((input_blocks_to_read - 1) / blockDim.x) + 1;

	unsigned pos = tid;

	for (unsigned i=0; i < readingLoops; i++) {
		if (pos < input_blocks_to_read) {
			shared_inputs[pos] = inputs[pos];
		}
		pos += blockDim.x;
	}
	__syncthreads();

	unsigned outputNeuron = blockIdx.x*blockDim.x + threadIdx.x;
	if (outputNeuron < output_size) {

		float result = 0;

		for (unsigned i=0; i < input_blocks_to_read; i++) {

			//TODO TCC check performance penalty (this is just for BT_SIGN)
			unsigned maxBits = min(BITS_PER_UNSIGNED, input_size - (i * BITS_PER_UNSIGNED));

			unsigned weighsOffset = (i * BITS_PER_UNSIGNED * output_size) + outputNeuron;
			unsigned input_block = shared_inputs[i];
			unsigned mask = 0x80000000;
			for (unsigned j=0; j < maxBits; j++) {

				if (input_block & mask) {
					result += weighs[weighsOffset] - 128;
				} else {
					if (inputType == BT_SIGN) {
						result += 128 - weighs[weighsOffset];
					}
				}
				weighsOffset += output_size;
				mask >>= 1;
			}
		}
		results[outputNeuron] += result;
	}
}

extern "C" void cuda_inputCalculation(void* inputPtr, unsigned input_size,
		BufferType inputType, unsigned output_size, void* weighs,
		float* results, unsigned block_size)
{
	unsigned grid_size = ((output_size - 1) / block_size) + 1;
	unsigned shared_mem_size;

	if (inputType == BT_BYTE) {
		std::string error = "cuda_inputCalculation is not implemented for BufferType BYTE as input.";
		throw error;
	}
	else if (inputType == BT_FLOAT) {
		if (input_size > 4032) {
			string error = "The maximum float input size is 4032.";
			throw error;
		}
		shared_mem_size = input_size * sizeof(float);

		SumFloatsConnectionsKernel<<< grid_size, block_size, shared_mem_size >>>((float*)inputPtr, input_size, output_size, (float*)weighs, results);
	} else {

		shared_mem_size =(((input_size - 1)/BITS_PER_UNSIGNED) + 1) * sizeof(unsigned);
		if (shared_mem_size > 16128) {
			//16128 * 8
			string error = "The maximum bit/sign input size is 129024.";
			throw error;
		}
		if (inputType == BT_BIT) {
			SumBitsConnectionsKernel<BT_BIT><<< grid_size, block_size, shared_mem_size >>>((unsigned*)inputPtr, input_size, output_size, (unsigned char*)weighs, results);
		} else {
			SumBitsConnectionsKernel<BT_SIGN><<< grid_size, block_size, shared_mem_size >>>((unsigned*)inputPtr, input_size, output_size, (unsigned char*)weighs, results);
		}
	}
}

extern "C" void cuda_inputCalculationInvertedMatrix(void* inputPtr, unsigned input_size,
		BufferType inputType, unsigned output_size, void* weighs,
		float* results, unsigned block_size)
{
	unsigned grid_size = ((output_size - 1) / block_size) + 1;
	unsigned shared_mem_size;

	if (inputType == BT_BYTE) {
		std::string error = "cuda_inputCalculation is not implemented for BufferType BYTE as input.";
		throw error;
	}
	else if (inputType == BT_FLOAT) {
		while (input_size > CUDA_MAX_SHARED_FLOATS) {

			shared_mem_size = CUDA_MAX_SHARED_FLOATS * sizeof(float);
			SumFloatsInvertedConnectionsKernel<<< grid_size, block_size, shared_mem_size >>>((float*)inputPtr, CUDA_MAX_SHARED_FLOATS, (float*)weighs, results, output_size);
			inputPtr = (void*) ((float*) inputPtr + CUDA_MAX_SHARED_FLOATS);
			weighs = (void*) ((float*) weighs + (CUDA_MAX_SHARED_FLOATS
					* output_size));
			input_size -= CUDA_MAX_SHARED_FLOATS;
		}
		shared_mem_size = input_size * sizeof(float);
		SumFloatsInvertedConnectionsKernel<<< grid_size, block_size, shared_mem_size >>>((float*)inputPtr, input_size, (float*)weighs, results, output_size);
	} else {
		//TODO TCC esta parte no funciona bien
		while (input_size > CUDA_MAX_SHARED_BITS) {

			shared_mem_size = CUDA_MAX_SHARED_FLOATS * sizeof(unsigned);
			// TODO TCC probar sin emulación
//			printf("grid_size %d, block_size %d, shared_mem_size %d \n", grid_size, block_size, shared_mem_size);
			if (inputType == BT_BIT) {
				SumBitsInvertedConnectionsKernel<BT_BIT><<< grid_size, block_size, shared_mem_size >>>((unsigned*)inputPtr, CUDA_MAX_SHARED_BITS, output_size, (unsigned char*)weighs, results);
			} else {
				SumBitsInvertedConnectionsKernel<BT_SIGN><<< grid_size, block_size, shared_mem_size >>>((unsigned*)inputPtr, CUDA_MAX_SHARED_BITS, output_size, (unsigned char*)weighs, results);
			}
			inputPtr = (void*)((float*)inputPtr + CUDA_MAX_SHARED_FLOATS);
			weighs = (void*)((float*)weighs + (CUDA_MAX_SHARED_BITS * output_size));
			input_size -= CUDA_MAX_SHARED_BITS;
		}
		shared_mem_size =(((input_size - 1)/BITS_PER_UNSIGNED) + 1) * sizeof(unsigned);
		// TODO TCC probar sin emulación
		//printf("grid_size %d, block_size %d, shared_mem_size %d \n", grid_size, block_size, shared_mem_size);
		if (inputType == BT_BIT) {
			SumBitsInvertedConnectionsKernel<BT_BIT><<< grid_size, block_size, shared_mem_size >>>((unsigned*)inputPtr, input_size, output_size, (unsigned char*)weighs, results);
		} else {
			SumBitsInvertedConnectionsKernel<BT_SIGN><<< grid_size, block_size, shared_mem_size >>>((unsigned*)inputPtr, input_size, output_size, (unsigned char*)weighs, results);
		}
	}
}

template <unsigned int blockSize, BufferType inputType>
__global__
void SumConnectionsKernel(void* inputPtr, unsigned input_size, unsigned output_size, void* weighs, float* results)
{
	extern __shared__ float sdata[];

	unsigned weighsOffset = (blockIdx.x * input_size);

	float result = 0;
	unsigned i = threadIdx.x;

	if (inputType == BT_FLOAT) {
		while (i < input_size) {
			result += ((float*)inputPtr)[i] * ((float*)weighs)[weighsOffset + i];
			i += blockDim.x;
		}
	} else {
		weighsOffset += threadIdx.x * BITS_PER_UNSIGNED;

		unsigned input_blocks_to_read = ((input_size - 1) / BITS_PER_UNSIGNED) + 1;
		while (i < input_blocks_to_read) {

			//TODO TCC check performance penalty (this is just for BT_SIGN)
			unsigned maxBits = min(BITS_PER_UNSIGNED, input_size - (i * BITS_PER_UNSIGNED));

			unsigned mask = 0x80000000;
			unsigned currentInput = ((unsigned*)inputPtr)[i];

			for (unsigned j=0; j < maxBits; j++) {

				if (currentInput & mask) {
					result += ((unsigned char*)weighs)[weighsOffset + j] - 128;
				} else {
					if (inputType == BT_SIGN) {
						result -= ((unsigned char*)weighs)[weighsOffset + j] - 128;
					}
				}
				mask >>= 1;
			}
			i += blockSize;
			weighsOffset += blockDim.x * BITS_PER_UNSIGNED;
		}
	}

	unsigned tid = threadIdx.x;
	sdata[tid] = result;
	__syncthreads();

	if (blockSize >= 512) {if (tid < 256) {sdata[tid] += sdata[tid + 256];}__syncthreads();}
	if (blockSize >= 256) {if (tid < 128) {sdata[tid] += sdata[tid + 128];}__syncthreads();}
	if (blockSize >= 128) {if (tid < 64) {sdata[tid] += sdata[tid + 64];}__syncthreads();}

#if __DEVICE_EMULATION__
	if (blockSize >= 64) {if (tid < 32) {sdata[tid] += sdata[tid + 32];}__syncthreads();}
	if (blockSize >= 32) {if (tid < 16) {sdata[tid] += sdata[tid + 16];}__syncthreads();}
	if (blockSize >= 16) {if (tid < 8) {sdata[tid] += sdata[tid + 8];}__syncthreads();}
	if (blockSize >= 8) {if (tid < 4) {sdata[tid] += sdata[tid + 4];}__syncthreads();}
	if (blockSize >= 4) {if (tid < 2) {sdata[tid] += sdata[tid + 2];}__syncthreads();}
	if (blockSize >= 2) {if (tid < 1) {sdata[tid] += sdata[tid + 1];}__syncthreads();}
#else
	if (tid < 32) {
		if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
		if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
		if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
		if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
		if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
		if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
	}
#endif
	if (tid == 0) {
		results[blockIdx.x] += sdata[0];
	}
}

extern "C" void cuda_inputCalculationReduction(void* inputPtr, unsigned input_size, BufferType inputType, unsigned output_size, void* weighs,
		float* results, unsigned block_size)
{
	unsigned grid_size = output_size;
	unsigned shared_mem_size = block_size * sizeof(float);

	if (inputType == BT_BYTE) {
		std::string error = "cuda_inputCalculation is not implemented for BufferType BYTE as input.";
		throw error;
	}
	else if (inputType == BT_FLOAT) {
		switch (block_size) {
		case 512:
			SumConnectionsKernel<512, BT_FLOAT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 256:
			SumConnectionsKernel<256, BT_FLOAT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 128:
			SumConnectionsKernel<128, BT_FLOAT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 64:
			SumConnectionsKernel< 64, BT_FLOAT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 32:
			SumConnectionsKernel< 32, BT_FLOAT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 16:
			SumConnectionsKernel< 16, BT_FLOAT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 8:
			SumConnectionsKernel< 8, BT_FLOAT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 4:
			SumConnectionsKernel< 4, BT_FLOAT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 2:
			SumConnectionsKernel< 2, BT_FLOAT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 1:
			SumConnectionsKernel< 1, BT_FLOAT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		}
	} else if (inputType == BT_BIT) {
		switch (block_size) {
		case 512:
			SumConnectionsKernel<512, BT_BIT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 256:
			SumConnectionsKernel<256, BT_BIT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 128:
			SumConnectionsKernel<128, BT_BIT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 64:
			SumConnectionsKernel< 64, BT_BIT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 32:
			SumConnectionsKernel< 32, BT_BIT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 16:
			SumConnectionsKernel< 16, BT_BIT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 8:
			SumConnectionsKernel< 8, BT_BIT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 4:
			SumConnectionsKernel< 4, BT_BIT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 2:
			SumConnectionsKernel< 2, BT_BIT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 1:
			SumConnectionsKernel< 1, BT_BIT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		}
	} else {
		switch (block_size) {
		case 512:
			SumConnectionsKernel<512, BT_SIGN><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 256:
			SumConnectionsKernel<256, BT_SIGN><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 128:
			SumConnectionsKernel<128, BT_SIGN><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 64:
			SumConnectionsKernel< 64, BT_SIGN><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 32:
			SumConnectionsKernel< 32, BT_SIGN><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 16:
			SumConnectionsKernel< 16, BT_SIGN><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 8:
			SumConnectionsKernel< 8, BT_SIGN><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 4:
			SumConnectionsKernel< 4, BT_SIGN><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 2:
			SumConnectionsKernel< 2, BT_SIGN><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		case 1:
			SumConnectionsKernel< 1, BT_SIGN><<< grid_size, block_size, shared_mem_size >>>(inputPtr, input_size, output_size, weighs, results); break;
		}
	}
	checkCUDAError("cuda_inputCalculation2");
}

