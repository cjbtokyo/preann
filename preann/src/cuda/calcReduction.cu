#include "hip/hip_runtime.h"
#include "cudaCommon.h"

template <unsigned int blockSize, BufferType inputType>
__global__
void ReductionKernel(void* inputPtr, void* weighs, float* results, unsigned input_size)
{
    extern __shared__ float sdata[];

    unsigned weighsOffset = (blockIdx.x * input_size);

    float result = 0;
    unsigned i = threadIdx.x;

    if (inputType == BT_FLOAT) {
        while (i < input_size) {
            result += ((float*)inputPtr)[i] * ((float*)weighs)[weighsOffset + i];
            i += blockSize;
        }
    } else {
        weighsOffset += threadIdx.x * BITS_PER_UNSIGNED;

        unsigned input_blocks_to_read = ((input_size - 1) / BITS_PER_UNSIGNED) + 1;
        while (i < input_blocks_to_read) {

            unsigned maxBits = device_min(BITS_PER_UNSIGNED, input_size - (i * BITS_PER_UNSIGNED));

            unsigned mask = 0x80000000;
            unsigned currentInput = ((unsigned*)inputPtr)[i];

            for (unsigned j=0; j < maxBits; j++) {

                if (currentInput & mask) {
                    result += ((unsigned char*)weighs)[weighsOffset + j] - 128;
                } else {
                    if (inputType == BT_SIGN) {
                        result -= ((unsigned char*)weighs)[weighsOffset + j] - 128;
                    }
                }
                mask >>= 1;
            }
            i += blockSize;
            weighsOffset += blockSize * BITS_PER_UNSIGNED;
        }
    }
    __syncthreads();

    unsigned tid = threadIdx.x;
    sdata[tid] = result;
    __syncthreads();

    if (blockSize >= 512) {if (tid < 256) {sdata[tid] += sdata[tid + 256];}__syncthreads();}
    if (blockSize >= 256) {if (tid < 128) {sdata[tid] += sdata[tid + 128];}__syncthreads();}
    if (blockSize >= 128) {if (tid < 64) {sdata[tid] += sdata[tid + 64];}__syncthreads();}

#if __DEVICE_EMULATION__
    if (blockSize >= 64) {if (tid < 32) {sdata[tid] += sdata[tid + 32];}__syncthreads();}
    if (blockSize >= 32) {if (tid < 16) {sdata[tid] += sdata[tid + 16];}__syncthreads();}
    if (blockSize >= 16) {if (tid < 8) {sdata[tid] += sdata[tid + 8];}__syncthreads();}
    if (blockSize >= 8) {if (tid < 4) {sdata[tid] += sdata[tid + 4];}__syncthreads();}
    if (blockSize >= 4) {if (tid < 2) {sdata[tid] += sdata[tid + 2];}__syncthreads();}
    if (blockSize >= 2) {if (tid < 1) {sdata[tid] += sdata[tid + 1];}__syncthreads();}
#else
    if (tid < 32) {
        if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
        if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
        if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
        if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
        if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
        if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
    }
#endif

    if (tid == 0) {
        results[blockIdx.x] += sdata[0];
    }
}

extern "C" void cuda_netCalcReduction(BufferType inputType, unsigned block_size, void* inputPtr, void* weighs,
                                      float* results, unsigned input_size, unsigned output_size)
{
    unsigned grid_size = output_size;
    unsigned shared_mem_size = block_size * sizeof(float);

    if (inputType == BT_BYTE) {
        std::string error = "cuda_inputCalculation is not implemented for BufferType BYTE as input.";
        throw error;
    } else if (inputType == BT_FLOAT) {
        switch (block_size) {
            case 512:
                ReductionKernel<512, BT_FLOAT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 256:
                ReductionKernel<256, BT_FLOAT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 128:
                ReductionKernel<128, BT_FLOAT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 64:
                ReductionKernel< 64, BT_FLOAT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 32:
                ReductionKernel< 32, BT_FLOAT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 16:
                ReductionKernel< 16, BT_FLOAT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 8:
                ReductionKernel<  8, BT_FLOAT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 4:
                ReductionKernel<  4, BT_FLOAT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 2:
                ReductionKernel<  2, BT_FLOAT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 1:
                ReductionKernel<  1, BT_FLOAT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
        }
    } else if (inputType == BT_BIT) {
        switch (block_size) {
            case 512:
                ReductionKernel<512, BT_BIT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 256:
                ReductionKernel<256, BT_BIT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 128:
                ReductionKernel<128, BT_BIT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 64:
                ReductionKernel< 64, BT_BIT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 32:
                ReductionKernel< 32, BT_BIT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 16:
                ReductionKernel< 16, BT_BIT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 8:
                ReductionKernel<  8, BT_BIT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 4:
                ReductionKernel<  4, BT_BIT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 2:
                ReductionKernel<  2, BT_BIT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 1:
                ReductionKernel<  1, BT_BIT><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
        }
    } else {
        switch (block_size) {
            case 512:
                ReductionKernel<512, BT_SIGN><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 256:
                ReductionKernel<256, BT_SIGN><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 128:
                ReductionKernel<128, BT_SIGN><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 64:
                ReductionKernel< 64, BT_SIGN><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 32:
                ReductionKernel< 32, BT_SIGN><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 16:
                ReductionKernel< 16, BT_SIGN><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 8:
                ReductionKernel<  8, BT_SIGN><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 4:
                ReductionKernel<  4, BT_SIGN><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 2:
                ReductionKernel<  2, BT_SIGN><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
            case 1:
                ReductionKernel<  1, BT_SIGN><<< grid_size, block_size, shared_mem_size >>>(inputPtr, weighs, results, input_size); break;
        }
    }
    checkCUDAError("cuda_inputCalculation2");
}

